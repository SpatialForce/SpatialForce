//  Copyright (c) 2024 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "graph.h"
#include "device.h"

namespace vox {
Graph::Graph(const Device &device, hipGraphExec_t graph) : device_{device}, graph_{graph} {}

Graph::~Graph() {
    ContextGuard guard(device_.primary_context());
    check_cuda(hipGraphExecDestroy(graph_));
}

void Graph::launch() {
    ContextGuard guard(device_.primary_context());
    check_cuda(hipGraphLaunch(graph_, device_.stream.handle()));
}

void capture_begin(uint32_t index) {
    const auto &d = device(index);
    ContextGuard guard(d.primary_context());
    check_cuda(hipStreamBeginCapture(d.stream.handle(), hipStreamCaptureModeGlobal));
}

Graph capture_end(uint32_t index) {
    const auto &d = device(index);
    ContextGuard guard(d.primary_context());

    hipGraph_t graph = nullptr;
    check_cuda(hipStreamEndCapture(d.stream.handle(), &graph));

    // enable to create debug GraphVis visualization of graph
    // hipGraphDebugDotPrint(graph, "graph.dot", hipGraphDebugDotFlagsVerbose);

    hipGraphExec_t graph_exec = nullptr;
    // check_cuda(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));

    // can use after CUDA 11.4 to permit graphs to capture hipMallocAsync() operations
    check_cuda(hipGraphInstantiateWithFlags(&graph_exec, graph, hipGraphInstantiateFlagAutoFreeOnLaunch));

    // free source graph
    check_cuda(hipGraphDestroy(graph));

    return {d, graph_exec};
}

}// namespace vox