//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "event.h"

namespace vox {
Event::Event(Device &device, bool enable_timing) : device_{device} {
    ContextGuard guard(device_.primary_context());

    int flags = hipEventDefault;
    if (!enable_timing) {
        flags |= hipEventDisableTiming;
    }

    check_cu(hipEventCreateWithFlags(&event_, flags));
}

Event::~Event() {
    ContextGuard guard(device_.primary_context());
    check_cu(hipEventDestroy(event_));
}

}// namespace vox