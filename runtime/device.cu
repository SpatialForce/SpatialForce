//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "device.h"

namespace vox {
Device::Device() {
    if (!check_cu(hipInit(0))) return;

    int deviceCount = 0;
    if (check_cu(hipGetDeviceCount(&deviceCount))) {
        all_devices.resize(deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            hipDevice_t device;
            if (check_cu(hipDeviceGet(&device, i))) {
                // query device info
                all_devices[i].device = device;
                all_devices[i].ordinal = i;
                check_cu(hipDeviceGetName(all_devices[i].name, DeviceInfo::kNameLen, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_memory_pool_supported,
                                              hipDeviceAttributeMemoryPoolsSupported, device));
                int major = 0;
                int minor = 0;
                check_cu(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                all_devices[i].arch = 10 * major + minor;
            }
        }
    }
}

const Device::DeviceInfo &device_info(uint32_t index) {
    static Device cuda_device;
    return cuda_device.all_devices[index];
}

}// namespace vox