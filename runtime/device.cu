#include "hip/hip_runtime.h"
//  Copyright (c) 2024 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "device.h"
#include <map>

namespace vox {
// Dummy kernel for retrieving PTX version.
template<int dummy_arg>
__global__ void dummy_k() {}

void DeviceInfo::primary_context_retain() {
    check_cu(hipDevicePrimaryCtxRetain(&primary_context, handle));
}

static std::vector<DeviceInfo> all_devices;

void init() {
    hipInit(0);

    int deviceCount = 0;
    if (check_cu(hipGetDeviceCount(&deviceCount))) {
        all_devices.resize(deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            hipDevice_t device;
            if (check_cu(hipDeviceGet(&device, i))) {
                hipGetDeviceProperties(&all_devices[i].props, i);
                hipFuncAttributes attr{};
                check_cuda(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(dummy_k<0>)));
                all_devices[i].ptx_version = attr.ptxVersion;

                // query device info
                all_devices[i].handle = device;
                all_devices[i].ordinal = i;
                check_cu(hipDeviceGetName(all_devices[i].name, DeviceInfo::kNameLen, device));
                check_cu(hipDeviceGetUuid(&all_devices[i].uuid, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].pci_domain_id, hipDeviceAttributePciDomainId, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].pci_bus_id, hipDeviceAttributePciBusId, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].pci_device_id, hipDeviceAttributePciDeviceId, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_memory_pool_supported, hipDeviceAttributeMemoryPoolsSupported, device));
                int major = 0;
                int minor = 0;
                check_cu(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                all_devices[i].arch = 10 * major + minor;
                all_devices[i].primary_context_retain();
            }
        }
    }
}

size_t device_count() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
}

const Device &device(uint32_t index) {
    static std::map<hipDevice_t, Device> cuda_device;

    auto handle = all_devices[index].handle;
    auto it = cuda_device.find(handle);
    if (it != cuda_device.end()) {
        return it->second;
    } else {
        auto result = cuda_device.emplace(handle, Device(&all_devices[index]));
        return result.first->second;
    }
}

void synchronize(uint32_t index) {
    auto &d = device(index);
    ContextGuard guard(d.primary_context());
    check_cu(hipCtxSynchronize());
}

//---------------------------------------------------------------------------------------------------------------
Device::Device(DeviceInfo *info)
    : _info{info},
      null_stream(*this, nullptr),
      stream(*this) {
}

const DeviceInfo &Device::info() const {
    return *_info;
}

hipCtx_t Device::primary_context() const {
    return _info->primary_context;
}

}// namespace vox