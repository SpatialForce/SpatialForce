//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "device.h"
#include <cassert>

namespace vox {
class DeviceInfo {
public:
    // cached info for all devices, indexed by ordinal
    std::vector<Device> all_devices;

    DeviceInfo();
};

DeviceInfo::DeviceInfo() {
    if (!check_cu(hipInit(0))) return;

    int deviceCount = 0;
    if (check_cu(hipGetDeviceCount(&deviceCount))) {
        all_devices.resize(deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            hipDevice_t device;
            if (check_cu(hipDeviceGet(&device, i))) {
                // query device info
                all_devices[i].device = device;
                all_devices[i].ordinal = i;
                check_cu(hipDeviceGetName(all_devices[i].name, Device::kNameLen, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_memory_pool_supported,
                                              hipDeviceAttributeMemoryPoolsSupported, device));
                int major = 0;
                int minor = 0;
                check_cu(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                all_devices[i].arch = 10 * major + minor;
            }
        }
    }
}

size_t device_count() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
}

const Device &device(uint32_t index) {
    static DeviceInfo cuda_device;
    assert(index < cuda_device.all_devices.size());
    return cuda_device.all_devices[index];
}

}// namespace vox