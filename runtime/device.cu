//  Copyright (c) 2023 Feng Yang
//
//  I am making my contributions/submissions to this project solely in my
//  personal capacity and am not conveying any rights to any intellectual
//  property of any third parties.

#include "device.h"
#include <cassert>

namespace vox {
class DeviceInfo {
public:
    // cached info for all devices, indexed by ordinal
    std::vector<Device> all_devices;

    DeviceInfo();
};

DeviceInfo::DeviceInfo() {
    int deviceCount = 0;
    if (check_cu(hipGetDeviceCount(&deviceCount))) {
        all_devices.resize(deviceCount);

        for (int i = 0; i < deviceCount; i++) {
            hipDevice_t device;
            if (check_cu(hipDeviceGet(&device, i))) {
                // query device info
                all_devices[i].handle = device;
                all_devices[i].ordinal = i;
                check_cu(hipDeviceGetName(all_devices[i].name, Device::kNameLen, device));
                check_cu(hipDeviceGetUuid(&all_devices[i].uuid, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].pci_domain_id, hipDeviceAttributePciDomainId, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].pci_bus_id, hipDeviceAttributePciBusId, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].pci_device_id, hipDeviceAttributePciDeviceId, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_uva, hipDeviceAttributeUnifiedAddressing, device));
                check_cu(hipDeviceGetAttribute(&all_devices[i].is_memory_pool_supported, hipDeviceAttributeMemoryPoolsSupported, device));
                int major = 0;
                int minor = 0;
                check_cu(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
                check_cu(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
                all_devices[i].arch = 10 * major + minor;

                all_devices[i]._primary_context_retain();
            }
        }
    }
}

void init() {
    hipInit(0);
}

size_t device_count() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
}

const Device &device(uint32_t index) {
    static DeviceInfo cuda_device;
    assert(index < cuda_device.all_devices.size());
    return cuda_device.all_devices[index];
}

void Device::_primary_context_retain() {
    check_cu(hipDevicePrimaryCtxRetain(&primary_context, handle));
}

}// namespace vox